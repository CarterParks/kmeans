#include<stdlib.h>
#include<stdio.h>
#include<string.h>
#include<math.h>
#include "kmeans.h"

int main(int argc, char *argv[]){
  double *data;
  hipMallocManaged(&data, COLS * ROWS * sizeof *data);
  int fs = table(argv[1], data);
  int *cluster = kmeans(data);
  for (int i = 0; i < ROWS; i++) printf("%d\n", cluster[i]);
  return 0;
}
