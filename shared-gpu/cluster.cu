#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include "kmeans.h"
#include <time.h>

__global__ void assign(int *clus, double *data, double *cent){
  int p = blockIdx.x;
  double min;
  for (int k = 0; k < K; k++){
    double sum = 0;
    for (int i = 0; i < COLS; i++){
      double p1 = data[p * COLS + i];
      double p2 = cent[k * COLS + i];
      sum += pow(p1 - p2, 2);
    }
    double s = sqrt(sum);
    if (s < min || k == 0){
      min = s;
      clus[p] = k;
    }
  }
}

__global__ void centroids(double *data, int *clus, double *cent){
  int k = blockIdx.x;
  int c = threadIdx.x;
  int n = 0;
  for (int p = 0; p < ROWS; p++)
    if (clus[p] == k){
      cent[k * COLS + c] += data[p * COLS + c];
      n++;
    }
  cent[k * COLS + c] /= n;
}

int *kmeans(double *data){
  double *cent;
  hipMallocManaged(&cent, K * COLS * sizeof (double));

  int *clus;
  hipMallocManaged(&clus, ROWS * sizeof (int));

  int *clus_p;
  hipMallocManaged(&clus_p, ROWS * sizeof (int));

  // initialize centroids
  int step = ROWS / K;
  int point = rand() % step;

  for (int i = 0; i < K; i++){
    memcpy(cent + i * COLS, data + point * COLS, COLS * sizeof *data);
    point = (point + step) % ROWS;
  }

  while (1) {
    assign<<<ROWS,1>>>(clus, data, cent);
    hipDeviceSynchronize();

    if (!memcmp(clus_p, clus, ROWS * sizeof (int))) break;
    hipMemcpy(clus_p, clus, ROWS * sizeof (int), hipMemcpyHostToHost);

    // calculate centroids
    centroids<<<K, COLS>>>(data, clus, cent);
    hipDeviceSynchronize();
  }

  return clus;
}
